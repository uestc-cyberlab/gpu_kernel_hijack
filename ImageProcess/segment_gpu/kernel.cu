#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

// Device code
extern "C" __global__ void kernel(unsigned char* A, unsigned char* B, int n) 
{
    	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int threshold = 230;
    	if(idx<n)
    	{
    		if(B[idx]>=threshold)
    		{
    			A[idx] = 255;
    		}
    		else
    		{
    			A[idx] = 0;
    		}
    	}
}

//nvcc -arch=sm_61 --cubin kernel.cu

