
#include <hip/hip_runtime.h>
﻿//nvcc -arch=sm_89 --cubin kernel.cu

__global__ void addKernel(int *c, int *a, int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
